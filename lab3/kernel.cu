﻿#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>

#define BLOCK_SIZE 1024
#define RADIUS 4
#define LENGTH 1024

__global__ void filter(const float* a, float* b, int length) {
	__shared__ float memory[BLOCK_SIZE + 2 * RADIUS];

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int localIndex = threadIdx.x + RADIUS;

	memory[localIndex] = a[threadIndex];

	if (threadIdx.x < RADIUS) {
		if (threadIndex - RADIUS < 0) {
			memory[localIndex - RADIUS] = a[length + threadIndex - RADIUS];
		}
		else {
			memory[localIndex - RADIUS] = a[threadIndex - RADIUS];
		}

		if (threadIndex + BLOCK_SIZE >= LENGTH) {
			memory[localIndex + BLOCK_SIZE] = a[threadIndex + BLOCK_SIZE - length];
		}
		else {
			memory[localIndex + BLOCK_SIZE] = a[threadIndex + BLOCK_SIZE];
		}
	}

	__syncthreads();

	float result = 0.0;

	for (int offset = -RADIUS; offset <= RADIUS; offset++) {
		result += memory[localIndex + offset];
	}

	b[threadIndex] = result;
}

int main() {
	float* x = (float*)malloc(sizeof(float) * LENGTH);
	float* y = (float*)malloc(sizeof(float) * LENGTH);

	for (int i = 0; i < LENGTH; ++i) {
		x[i] = 1.0f;
		y[i] = -1.0f;
	}

	float* d_x, * d_y;
	hipMalloc(&d_x, sizeof(float) * LENGTH);
	hipMalloc(&d_y, sizeof(float) * LENGTH);

	hipMemcpy(d_x, x, sizeof(float) * LENGTH, hipMemcpyHostToDevice);

	int blocksNum = (LENGTH + BLOCK_SIZE - 1) / BLOCK_SIZE;

	filter<<< blocksNum, BLOCK_SIZE >>>(d_x, d_y, LENGTH);
	hipDeviceSynchronize();

	hipMemcpy(y, d_y, sizeof(float) * LENGTH, hipMemcpyDeviceToHost);

	printf("Idx\tX[i]\t\tY[i]\n");
	for (int i = 0; i < LENGTH; ++i) {
		printf("%d\t%.2f\t->\t%.2f\n", i, x[i], y[i]);
	}

	free(x);
	free(y);
	hipFree(d_x);
	hipFree(d_y);

	return 0;
}